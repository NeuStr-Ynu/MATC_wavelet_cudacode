#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);  // 获取设备数量

    if (deviceCount == 0) {
        std::cout << "没有找到CUDA支持的设备！" << std::endl;
        return 1;
    }

    for (int deviceId = 0; deviceId < deviceCount; deviceId++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, deviceId);  // 获取设备属性

        std::cout << "设备 " << deviceId << "： " << deviceProp.name << std::endl;

        // 查看每个设备的流处理单元数量（CUDA核心数量）
        std::cout << "每个流处理单元的最大线程数: " << deviceProp.maxThreadsPerBlock << std::endl;

        // 查看每个多处理器（Streaming Multiprocessor，SM）的数量
        std::cout << "流处理单元数量 (Multiprocessor count): " << deviceProp.multiProcessorCount << std::endl;

        // 计算最大线程数
        int maxThreadsPerGrid = deviceProp.maxGridSize[0];  // 最大网格维度
        std::cout << "最大网格大小: " << maxThreadsPerGrid << std::endl;
        std::cout << "每个块的最大线程数: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "每个块的最大维度：[" << deviceProp.maxThreadsDim[0] << ", "
            << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << "]" << std::endl;
    }

    return 0;
}
