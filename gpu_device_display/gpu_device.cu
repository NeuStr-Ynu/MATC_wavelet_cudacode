#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);  // ��ȡ�豸����

    if (deviceCount == 0) {
        std::cout << "û���ҵ�CUDA֧�ֵ��豸��" << std::endl;
        return 1;
    }

    for (int deviceId = 0; deviceId < deviceCount; deviceId++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, deviceId);  // ��ȡ�豸����

        std::cout << "�豸 " << deviceId << "�� " << deviceProp.name << std::endl;

        // �鿴ÿ���豸��������Ԫ������CUDA����������
        std::cout << "ÿ��������Ԫ������߳���: " << deviceProp.maxThreadsPerBlock << std::endl;

        // �鿴ÿ���ദ������Streaming Multiprocessor��SM��������
        std::cout << "������Ԫ���� (Multiprocessor count): " << deviceProp.multiProcessorCount << std::endl;

        // ��������߳���
        int maxThreadsPerGrid = deviceProp.maxGridSize[0];  // �������ά��
        std::cout << "��������С: " << maxThreadsPerGrid << std::endl;
        std::cout << "ÿ���������߳���: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "ÿ��������ά�ȣ�[" << deviceProp.maxThreadsDim[0] << ", "
            << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << "]" << std::endl;
    }

    return 0;
}
