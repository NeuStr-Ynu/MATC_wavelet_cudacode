#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include "wavelet.cuh"

__constant__ double sqrt_2 = 1.414213562373095;

__global__ void haar1Dkernel(double* input, double* output, int signalLength)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= signalLength / 2) return;


    output[idx] = (input[2 * idx] + input[2 * idx + 1]) / sqrt_2; // Approximation
    output[idx + signalLength / 2] = (input[2 * idx] - input[2 * idx + 1]) / sqrt_2; // Detail
}

void haar1D_gpu(double* input, double* output, int signalLength) {
    // ����źų����Ƿ�Ϊ2���ݴη�
    if ((signalLength & (signalLength - 1)) != 0) {
        std::cerr << "Signal length must be a power of 2." << std::endl;
        return;
    }


    double* d_input;
    double* d_output;

    hipMalloc(&d_input, signalLength * sizeof(double));
    hipMalloc(&d_output, signalLength * sizeof(double));

    hipMemcpy(d_input, input, signalLength * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (signalLength / 2 + threadsPerBlock - 1) / threadsPerBlock;

    haar1Dkernel <<< blocksPerGrid, threadsPerBlock >>> (d_input, d_output, signalLength);
    hipGetLastError();
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, signalLength * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void fullDecomposition(double* input, double* output, int signalLength) {

    int levels = log2(signalLength); // ������ȫ�ֽ�Ĳ���
    double* tempInput = input;
    double* tempOutput = output;

    for (int i = 0; i < levels; i++) {
        int currentLength = signalLength >> i; // ��ǰ���źŵĳ���
        haar1D_gpu(tempInput, tempOutput, currentLength);

        // ��������������루ֻ���ǰ�벿�ֽ���ϵ����
        tempInput = tempOutput; // ����Ϊ��һ�������
    }
}
