﻿#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "MATC_linker.h"
#include "wavelet.cuh"

/*Written by Zipeng Wang @KunMing*/

int main() {
    int size_of = sizeof(double);

    void* voidptr_data = MapFileToMemory();  // Ensure this works
    if (voidptr_data == nullptr) {
        std::cout << "[error]: Data does not exist." << std::endl;
        return 1;
    }

    int dataLength = GetInputFileSize(INPUT_FILENAME) / size_of;

    double* data_ptr = (double*)malloc(dataLength * sizeof(double));
    if (data_ptr == NULL) {
        std::cout << "[error]: Memory allocation failed!\n" << std::endl;
        return 1;
    }
    double* result_ptr = (double*)malloc(dataLength * sizeof(double));
    std::memcpy(data_ptr, voidptr_data, dataLength * sizeof(double));

    std::cout << "First ten inpute data:" << std::endl;
    for (int i = 0; i < 10 && i < dataLength; ++i) {
        std::cout << data_ptr[i] << std::endl;
    }


    // 计算 haar1D_gpu
    haar1D_gpu(data_ptr, result_ptr, dataLength);

    // 确保CUDA完成后再继续
    hipDeviceSynchronize();

    std::cout << "First ten transformed results:" << std::endl;
    for (int i = 0; i < 10 && i < dataLength; ++i) {
        std::cout << result_ptr[i] << std::endl;
    }

    // 输出数据
    FILE* file = fopen("outdata.bin", "wb");
    if (file == NULL) {
        std::cout << "[error]: outdata.bin Failed to open file\n" << std::endl;
        free(data_ptr);
        free(result_ptr);
        return 1;
    }

    if (fwrite(result_ptr, sizeof(double), dataLength, file) != dataLength) {
        perror("Failed to write data to file");
        fclose(file);
        free(data_ptr);
        free(result_ptr);
        return 1;
    }

    // 关闭文件
    fclose(file);

    // 释放内存
    free(data_ptr);
    free(result_ptr);

    return 0;
}
